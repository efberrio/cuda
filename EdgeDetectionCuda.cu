#include <iostream>
#include <math.h>
#include <fstream>
#include <vector>
#include <sstream>
#include <time.h>
#include <stdlib.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;

const int THREADS_PER_BLOCK = 512;

/**
 * CUDA Kernel Device code
 */ 
/*****************************************************************************/

__global__ void scaleImageCuda (int *pixels, int minpix, int maxpix, int imageSize) {
	/* blockDim.x gives the number of threads per block, combining it
	with threadIdx.x and blockIdx.x gives the index of each global
	thread in the device */
	int index = threadIdx.x * blockIdx.x * threadIdx.x;
	int value;
	/* Typical problems are not friendly multiples of blockDim.x.
	Avoid accesing data beyond the end of the arrays */
	if (index < imageSize) {
		value = round(((double)(pixels[index] - minpix) / (maxpix - minpix)) * 255);
		pixels[index] = value;
	}

    __syncthreads();
}

__global__ void edgeDetectionCuda (int *pixels, int *tempImage, int width, int height, int imageSize) {
	/* blockDim.x gives the number of threads per block, combining it
	with threadIdx.x and blockIdx.x gives the index of each global
	thread in the device */
	//int index = threadIdx.x * blockIdx.x * threadIdx.x;
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	int x = 0, y = 0;
	int xG = 0, yG = 0;

	/* Typical problems are not friendly multiples of blockDim.x.
	Avoid accesing data beyond the end of the arrays */
	if (index < imageSize) {
		x = index % width;

		if (index != 0) {
			y = __double2int_rd((__int2double_rn(index) / __int2double_rn(width)));	
		}
		
		if (x < (width - 1) && y < (height - 1)
				&& (y > 0) && (x > 0)) {

			//index = x + (y * width)
			//Finds the horizontal gradient
			xG = (pixels[(x+1) + ((y-1) * width)]
						 + (2 * pixels[(x+1) + (y * width)])
						 + pixels[(x+1) + ((y+1) * width)]
								  - pixels[(x-1) + ((y-1) * width)]
										   - (2 * pixels[(x-1) + (y * width)])
										   - pixels[(x-1) + ((y+1) * width)]);

			//Finds the vertical gradient
			yG = (pixels[(x-1) + ((y+1) * width)]
						 + (2 * pixels[(x) + ((y + 1) * width)])
						 + pixels[(x+1) + ((y+1) * width)]
								  - pixels[(x-1) + ((y-1) * width)]
										   - (2 * pixels[(x) + ((y-1) * width)])
										   - pixels[(x+1) + ((y-1) * width)]);
			tempImage[index] = __double2int_rn(sqrt(__int2double_rn(xG * xG) + __int2double_rn(yG * yG)));

		} else {

			//Pads out of bound pixels with 0
			tempImage[index] = 0;

		}
	}

    //__syncthreads();
}

//Creating image class (base class)
class Image{

public:

	Image():
		height(0),
		width(0),
		maxPixelValue(0),
		minpix(0),
		maxpix(0),
		imageSize(0){}
	virtual ~Image(){}

	virtual void readImage(ifstream &inFile) = 0;
	virtual void writeImage(ofstream &outFile) = 0;

	void readHeader(ifstream &inFile);
	void scaleImage();
	void edgeDection();

	//Accessor methods
	int getHeight(){return height;}
	int getWidth(){return width;}
	int getMaxPixelValue(){return maxPixelValue;}

	//Mutator methods
	void setHeight(int h){height = h;}
	void setWidth(int w){width = w;}
	void setMaxPixelValue(int mpv){maxPixelValue = mpv;}

	//Member variables
protected:

	int height;
	int width;
	int maxPixelValue;
	int minpix;
	int maxpix;
	unsigned int imageSize;
	int * pixels;

	inline void findMin();
	inline void findMax();

};

//Binary image class (derived class)

class BinaryImage: public Image{

public:

	BinaryImage(){}
	~BinaryImage(){}

	void readImage(ifstream &inFile);
	void writeImage(ofstream &outFile);

};

class AsciiImage: public Image{

public:

	AsciiImage(){}
	~AsciiImage(){}

	void readImage(ifstream &infile);
	void writeImage(ofstream &outFile);

};

//Check if header contains comments
//Comments start with #
bool isComment(string comment){

	for(unsigned int i = 0; i < comment.length(); i++){

		if(comment[i] == '#') return true;

		if(!isspace(comment[i])) return false;

	}

	return true;
}

//Reads binary pixel values in image
void BinaryImage::readImage(ifstream &inFile){

	//Check if the file stream in open
	if(!inFile){

		cerr << "Could not read from file!" << endl;

		exit(1000);

	}

	//Making a temp array, and putting it on the heap
	char * byteArray = new char[imageSize + 1];

	//Read the bytes of the image, and puts data in byteArray
	inFile.read(byteArray, imageSize);

	//If reading in the data failed, return an error
	if(inFile.fail()){

		cerr << "Error: cannot read pixels." << endl;

		exit(1000);

	}

	//Set the last element in array to EOF character
	byteArray[imageSize] = '\0';

	//Put the data read from file into pixels
	pixels = (int *)malloc(imageSize * sizeof(int));
	for(unsigned int i = 0; i < imageSize; i++){

		pixels[i] = static_cast<int>
		(static_cast<unsigned char>(byteArray[i]));

	}

	//Delete the byteArray
	free(byteArray);

}

//Writes binary pixels to output file
void BinaryImage::writeImage(ofstream &outFile){

	//Check if the file stream is open
	if(!outFile){

		cerr << "Could not write to file." << endl;

		exit(1000);

	}

	//Write header
	outFile << "P5"       << " "  <<
			width         << " "  <<
			height        << " "  <<
			maxPixelValue << endl;

	//Take all pixel values from pixels and writes it to output file
	char * byteArray = new char[imageSize + 1];

	for(unsigned int i = 0; i < imageSize; i++){

		byteArray[i] = static_cast<char>(pixels[i]);

	}

	byteArray[imageSize] = '\0';

	outFile.write(byteArray, imageSize);

	if(outFile.fail()){

		cerr << "Error: error writing to file." << endl;

		exit(1000);

	}

	free(byteArray);
	free(pixels);
	//delete[] byteArray;

}

void AsciiImage::readImage(ifstream &inFile){

	//Check if the file opened properly
	if(!inFile){

		cerr << "Could not read from file." << endl;

		exit(1001);

	}

	int pixelValue;

	//Read in the Ascii values from file
	int i = 0;
	while(inFile >> pixelValue){

		pixels[i] = pixelValue;
		i++;

	}


}


void AsciiImage::writeImage(ofstream &outFile){

	//Check if file is open
	if(!outFile){

		cerr << "Could not write to file." << endl;

		exit(1001);

	}

	//Write Header
	outFile << "P2" << ' ' <<
			width << ' ' <<
			height << ' ' <<
			maxPixelValue << '\n';

	//Write the contents of pixels to the output file
	for(unsigned int i = 0; i < imageSize; i++){

		//Add a '\n' at the end of each row
		if(i % width == 0 && i != 0) outFile << '\n';

		outFile << pixels[i] << '\t';

	}

	free(pixels);
}

void Image::readHeader(ifstream &inFile){

	stringstream sStream;

	string line;

	//Check if the file opened successfully
	if(!inFile){

		cerr << "Error: Could not open file." << endl;

		exit(1002);

	}

	char readChar;

	string errorMessage = "Error: incorrect picture format.";

	getline(inFile, line);

	unsigned int lineSize = line.length();

	//After we read magic number, we read the next line and determine if it's valid
	for(unsigned int i = 0; i < lineSize; i++){

		if(!isspace(line[i])){

			cerr << errorMessage << endl;

			cerr << "Extra info after magic number." << endl;

			exit(1002);

		}

	}

	//Read through the rest of the header and skip through comments
	while(getline(inFile, line)){

		if(!(isComment(line))) break;

	}

	sStream << line;

	//Read in width.
	//If there is a problem, return error
	if(!(sStream >> width)){

		cerr << errorMessage << endl;

		cerr << "Cannot read width." << endl;

		exit(1002);

	}

	//Read in height
	//If there is a problem, return error
	if(!(sStream >> height)){

		cerr << errorMessage << endl;

		cerr << "Cannot read height." << endl;

		exit(1002);

	}

	//Check if there is extra information after width and height
	while(sStream >> readChar){

		if(!(isspace(readChar))){

			cerr << errorMessage << endl;

			cerr << "Extra info when reading height and width." << endl;

			exit(1002);

		}

	}

	//Make sure the height and width is positive
	if(width <= 0 || height <= 0){

		cerr << "Error: width and height cannot be negative" << endl;

		exit(1002);

	}

	//Check if there are any comments between height/width and maxPixelValue
	while(getline(inFile, line)){

		if(!(isComment(line))) break;

	}

	//Clear out the string stream
	sStream.str("");
	sStream.clear();

	sStream << line;

	//Read in the maxPixelValue
	if(!(sStream >> maxPixelValue)){

		cerr << errorMessage << endl;
		cerr << "Could not read maxPixelValue." << endl;

		exit(1002);

	}

	//Check if there is extra information after maxPixelValue
	while(sStream >> readChar){

		if(!(isspace(readChar))){

			cerr << errorMessage << endl;
			cerr << "Extra info after the max pixel value." << endl;

			exit(1002);

		}

	}

	if(maxPixelValue < 0 || maxPixelValue > 255){

		cerr << errorMessage << endl;
		cerr << "Invalid max pixel value." << endl;

		exit(1002);

	}

	imageSize = width * height;

}

//Finds the maxium pixel value in the image
void Image::findMax(){

	int maxVal = 0;

	for(unsigned int i = 0; i < imageSize; i++){

		if(pixels[i] > maxVal){

			maxVal = pixels[i];

		}

	}

	maxpix = maxVal;

}

//Finds the minimal pixel value of the image
void Image::findMin(){

	int minVal = 255;

	for(unsigned int i = 0; i < imageSize; i++){

		if(pixels[i] < minVal){

			minVal = pixels[i];

		}

	}

	minpix = minVal;

}

//Scales image so that the maximum pixel value is 255
void Image::scaleImage(){

	findMin();

	findMax();

	printf("OK1\n");

	int *d_pixels;
	size_t size = imageSize * sizeof(int);
    hipError_t err = hipSuccess;
	printf("OK2\n");

	/* Allocate memory in device */
	err = hipMalloc((void **) &d_pixels, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector pixels (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK3\n");

	/* Copy data to device */
	err = hipMemcpy(d_pixels, pixels, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector pixels from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK4\n");

	/* Launch scaleImageCuda() kernel on device with N threads in N blocks */
	scaleImageCuda<<<(imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_pixels, minpix, maxpix, imageSize);
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch scaleImageCuda kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK5\n");

	/* Copy data to tohost device */
	err = hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector pixels from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK6\n");

	/* Clean-up */
	err = hipFree(d_pixels);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector pixels (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK7\n");

    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("OK8\n");

	maxPixelValue = 255;

}

//Sobel edge detection function - detects edges and draws an outline
void Image::edgeDection(){
    hipError_t err = hipSuccess;
	size_t size = imageSize * sizeof(int);
	/* Allocate memory in host */
	int *tempImage = (int *)malloc(size);

	int *d_pixels, *d_tempImage;
	/* Allocate memory in device */
	err = hipMalloc((void **) &d_pixels, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device array pixels (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMalloc((void **) &d_tempImage, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device array tempImage (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	printf("alojo memoria\n");

	/* Copy data to device */
	err = hipMemcpy(d_pixels, pixels, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy array pixels from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMemcpy(d_tempImage, tempImage, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy array tempImage from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("copio memoria a device\n");

	/* Launch edgeDetectionCuda() kernel on device with N threads in N blocks */
	edgeDetectionCuda<<<(imageSize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_pixels, d_tempImage, width, height, imageSize);
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch edgeDetectionCuda kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("ejecuto kernel\n");

	/* Copy data to host */ 
	err = hipMemcpy(tempImage, d_tempImage, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy array tempImage from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("copio memoria a host\n");

	/* Clean-up device */
	err = hipFree(d_tempImage);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free array vector tempImage (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipFree(d_pixels);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free array vector pixels (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("libero memoria\n");

	for(unsigned int i = 0; i < imageSize; i++){

		pixels[i] = tempImage[i];

	}
	printf("copio resultado\n");

	/* Clean-up host */
	free(tempImage);
}

bool isBinary(ifstream &inFile);

void run(char **argv);

int main(int argc, char **argv){

	if(argc != 3){

		cerr << "Usage: EdgeDetection imageName.pgm output.pgm";

		return 1;

	}

	//long start, end;

	//double total;

	//start = clock();

	run(argv);

	//end = clock();

	//total = (double)(end - start)/1000;

	//cout << "Execution time: " << total << endl;

	return 0;
}


bool isBinary(ifstream &inFile){

	char readChar = ' ';

	string errorMessage = "Error: incorrect picture format.";

	//If there is no character or the character is not equal to 'P'
	//then return an error
	if(!(inFile >> readChar) || ( readChar != 'P' )){

		cerr << errorMessage << endl;
		cerr << "P" << endl;

		exit(1002);

	}

	//If there is no character or the second character is not a 2 or 5
	//then return an error
	if(!(inFile >> readChar) || ( readChar != '2' && readChar != '5')){

		cerr << errorMessage << endl;
		cerr << readChar << endl;

		exit(1002);

	}

	if(readChar == '5') return true;

	return false;

}

void run(char **argv){

	ifstream inFile;

	inFile.open(argv[1], ios::binary | ios::in);

	ofstream outFile;

	outFile.open(argv[2], ios::binary
			            | ios::out
						| ios::trunc);

	if(isBinary(inFile)){

		BinaryImage binaryImage;

		binaryImage.readHeader(inFile);

		binaryImage.readImage(inFile);

		binaryImage.edgeDection();

		binaryImage.scaleImage();

		binaryImage.writeImage(outFile);

	}else{

		AsciiImage asciiImage;

		asciiImage.readHeader(inFile);

		asciiImage.readImage(inFile);

		asciiImage.edgeDection();

		asciiImage.scaleImage();

		asciiImage.writeImage(outFile);

	}

	inFile.close();
	outFile.close();

}


